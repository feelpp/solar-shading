#include "hip/hip_runtime.h"
#include "bvh.hpp"
#include "Bvh_GPU.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <device_atomic_functions.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <cfloat>


__host__ void checkCudaErrors(hipError_t error, const char* file, int line)
{
    if (error != hipSuccess)
    {
        printf("\nCUDA Error at %s:%d: %s\n", file, line, hipGetErrorString(error));
        exit(-1);
    }
}
#define CHECK_CUDA_ERRORS(call) checkCudaErrors((call), __FILE__, __LINE__)

struct GPURay
{
    float origin[3];
    float dir[3];

    GPURay() : origin{0.0f, 0.0f, 0.0f}, dir{0.0f, 0.0f, 0.0f} {}

    GPURay(const float o[3], const float d[3]) : origin{o[0], o[1], o[2]}, dir{d[0], d[1], d[2]} {}

    GPURay( const Feel::BVHRay R)
    {
        for (int i=0;i<3;i++)
        {
            origin[i]=R.origin[i];
            dir[i]=R.dir[i];
        }
    }
};

struct GPUNode
{
    GPUNode* parent;      // using pointers may cause enormous problems, since they are instanciated on the CPU, 
    GPUNode* leftchild;   // the pointers will still refer to the CPU memory, and not the GPU memory, so we could 
    GPUNode* rightchild;  // either use an array of indices ( for example giving the nodes indices during constructions),
    int nPrimitives;      // But since everythin is already implemented, we will use pointers, and implement a deep copy
    int firstPrimOffset;  // function that will copy the whole tree from the CPU to the GPU, and adjust the pointers
    int splitAxis;
    float bounds_min[3]; // one float per dimension
    float bounds_max[3];
    float centroid[3];

    GPUNode(const Feel::BVHTree<3>::BVHNode& bvhNode) 
    {
        parent = nullptr; // Parent needs to be set by whoever creates this node

        if (bvhNode.isLeaf()) 
        {
            buildLeaf(bvhNode.firstPrimOffset, 
                    bvhNode.nPrimitives, 
                    bvhNode.M_bounds_min.data(), 
                    bvhNode.M_bounds_max.data(), 
                    bvhNode.M_centroid.data());
        } 
        else 
        {
            this->splitaxis = bvhNode.splitaxis;
            if (bvhNode.children[0] != nullptr)
            {
                this->leftchild = new GPUNode(*(bvhNode.children[0]));
                this->leftchild->parent = this;
            }
            if (bvhNode.children[1] != nullptr) 
            {
                this->rightchild = new GPUNode(*(bvhNode.children[1]));
                this->rightchild->parent = this;
            }
            buildInternalNode(this->splitaxis,
                            this->leftchild,
                            this->rightchild);
        }
    }

    __host__ void buildLeaf(int first, int n, const float bmin[3], const float bmax[3], const float cent[3]) {
        firstPrimOffset = first;
        nPrimitives = n;
        leftchild = nullptr;
        rightchild = nullptr;
        for (int i=0; i<3; i++) {
            bounds_min[i] = bmin[i];
            bounds_max[i] = bmax[i];
            centroid[i] = cent[i];
        }
    }

    __host__ void buildInternalNode(int splitaxisIn, GPUNode* child0, GPUNode* child1) {
        leftchild = child0;
        rightchild = child1;
        splitAxis = splitaxisIn; 
        nPrimitives = 0;
        firstPrimOffset = -1;
        for (int i=0; i<3; i++) {
            bounds_min[i] = std::min(child0->bounds_min[i], child1->bounds_min[i]);
            bounds_max[i] = std::max(child0->bounds_max[i], child1->bounds_max[i]);
            centroid[i] = (bounds_min[i] + bounds_max[i]) / 2.0f;
        }
    }

        
    // Deep Copy recursive function
    __host__ GPUNode* DeepCopyToGPU() 
    {
        if (this == NULL)
        {
            return nullptr;
        }

        // Allocate memory for this node on the GPU
        GPUNode *node_copy;
        hipMalloc(&node_copy, sizeof(GPUNode));

        // Copy the node data to the GPU
        hipMemcpy(node_copy, this, sizeof(GPUNode), hipMemcpyHostToDevice);

        // Make recursive calls to copy children
        GPUNode *d_leftChild = nullptr;
        GPUNode *d_rightChild = nullptr;

        if (this->leftchild)
        {
            d_leftChild = this->leftchild->DeepCopyToGPU();
            hipMemcpy(&(node_copy->leftchild), &d_leftChild, sizeof(GPUNode*), hipMemcpyHostToDevice);
        }
        if (this->rightchild)
        {
            d_rightChild = this->rightchild->DeepCopyToGPU();
            hipMemcpy(&(node_copy->rightchild), &d_rightChild, sizeof(GPUNode*), hipMemcpyHostToDevice);
        }

        return node_copy;
    }



    // some methods for the traversal
    __device__ GPUNode * nearChild(GPURay const& ray)
    {
                
        if(ray.dir[this->splitaxis]>0)
            return this->child0;
        else
            return this->child1;

    }
            
    __device__ GPUNode * otherChild(GPUNode * parent)
    {
        if (this==parent->child0)
            return parent->child1;
         else
            return parent->child0;
    }

    __device__ bool checkIntersection(GPURay const& rayon)
    {
        float tmin = 0.0;
        float tmax = FLT_MAX;

        for(int i=0; i<nDim; i++)
        {                    
            float ratio = 1.0/(rayon.dir[i]+2*FLT_MIN);
            float t1 = (this->bounds_min[i]-rayon.origin[i]) * ratio;
            float t2 = (this->bounds_max[i]-rayon.origin[i]) * ratio;
            if (t1 > t2) 
            {
                float tTemp = t1;
                t1 = t2;
                t2 = tTemp;
            }
            if ( t1 > tmin)
                tmin = t1;
            if (t2 > tmax)
                tmax = t2;
            if (tmin > tmax)
                return false;
        }
        
        return true;         
    }

    __device__ bool isLeaf()
    {
        return (this->nPrimitives>0);
    }

    __device__ int getfirstPrimOffset()
    {
        return this->firstPrimOffset;
    }
};
        
struct GPUBVH
{
    GPUNode * M_root_gpu_tree;

    __host__ GPUNode * buildRootTree(Feel::BVHTree<3> * tree)
    {
        // Copy the necessary data from the BVHTree into the GPUBVH
        Feel::BHVTree::BVHNode * root_cpu_tree = tree->getRootNode();
        GPUNode * root_gpu_node = new GPUNode(root_cpu_tree);
        return root_gpu_node;
    }
    
    // this function only returns the first prim offset, we can link the first prim offset (stored in results)
    // to the ray which intersected it by using its index in the results array. If no intersection is found, the
    // array will display -2.
    __device__ void GPU_traverse_stackless(GPUNode * tree, GPURay const& ray, int * results, int & result_count)
    {
        auto current_node = tree -> nearChild(ray);
        char state = 'P'; 

        result_count = 0;       

        while (true)
        {
            switch (state)
            {
                case 'C':
                    if (current_node == M_root_gpu_tree) return;

                    if (current_node == current_node->parent->nearChild(ray))
                    {
                        current_node = current_node->otherChild(current_node->parent);
                        state = 'S'; // from Sibling
                    }
                    else 
                    {
                        current_node = current_node->parent;
                        state = 'C'; // the current node has been accessed from its sibling
                    }
                    break;

                case 'S': // the node is being traversed from its sibling
                    if (current_node->checkintersection(ray)==false) // go back to parent
                    {
                            current_node = current_node->parent;
                        state = 'C';
                    }
                    else if (current_node->isLeaf())
                    {
                        // either perform the Ray/Primitive intersection test here, or return 
                        // the primitive indices and let the CPU do the intersection tests
                        // if we pushback the firstPrimOffset, then we can use it to retrieve the primitive
                        // in the M_primitiveinfo array
                        int index = atomicAdd(&result_count, 1);            // Increment the result_count atomically and get the previous value as the index. This way, even if 
                        results[index] = current_node->getfirstPrimOffset();// multiple threads try to write to the same index, we will not lose any results since they'll be queued
                        current_node = current_node->parent;
                        state = 'C';
                    }
                    else
                    {
                        current_node = current_node->parent;
                        state = 'P';
                    }
                    break;
                
                case 'P':
                    if (current_node->checkIntersection(ray)==false)
                    {
                        current_node = current_node->otherChild(current_node->parent);
                        state = 'S';
                    }
                    else if (current_node->isLeaf())
                    {
                        int index = atomicAdd(&result_count, 1); // Increment the result_count atomically and get the previous value as the index
                        results[index] = current_node->getfirstPrimOffset();    
                        current_node = current_node->otherChild(current_node->parent);
                        state = 'S';
                    }
                    else
                    {
                        current_node = current_node->nearChild(ray);
                        state = 'P';
                    }
                    break;
                
                default:

                    break;
            }
        }
    }

    __global__ void GPU_traverse_kernel(GPUNode* tree, GPURay const* rays, int* results, int numRays)
    {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        int N = 10 ; // this is the max number of intersections we can have per ray
        if (index < numRays)
        {
            int thread_results[N]; // Local array specific to each thread
            int result_count = 0; // Local variable specific to each thread
            GPU_traverse_stackless(tree, rays[index], thread_results, result_count);

            for (int i = 0; i < result_count; i++)
            {
                results[index * 10 + i] = thread_results[i]; // Store results in the global results array
            }
        }
    }

    __host__ std::vector<int> GPUraySearch(std::vector<Feel::BVHRay> const& rays, const Feel::BVHTree<3> * tree)
    {
        int totalRays = rays.size();
        int numDevices;
        CHECK_CUDA_ERRORS(hipGetDeviceCount(&numDevices));
        int raysPerDevice = totalRays / numDevices; // Assuming totalRays is divisible by numDevices here.
        std::vector<double> lengths; // no distances are computed on the GPU
        std::vector<GPURay> rayons;
        std::vector<int> results(totalRays, -2);

        // convert the BVHRays to GPURays here
        for (int i = 0; i < totalRays; i++)
        {
            rayons.push_back(GPURay(rays[i]));
        }

        // Get all informations on the devices needed to perform the ray search
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);// Assumes all devices are identical
        int maxThreadsDim = prop.maxThreadsDim[0];  
        int maxGridSize = prop.maxGridSize[0];
        int maxThreadsPerBlock = prop.maxThreadsPerBlock;
        size_t totalGlobalMem = prop.totalGlobalMem;// Total global memory (in bytes) ===> can be used to check whether the tree fits in the GPU memory
        int threadsPerBlock = std::min(totalRays, maxThreadsPerBlock);
        int blocks = (totalRays + threadsPerBlock - 1) / threadsPerBlock; 
        int blocksPerGrid = (raysPerDevice + threadsPerBlock - 1) / threadsPerBlock; // Round up division

        M_root_gpu_tree = buildRootTree(tree);


        hipStream_t stream[numDevices];
        GPUNode *d_tree[numDevices];
        GPURay *d_rays[numDevices];
        int *d_results[numDevices];

        for (int i = 0; i < numDevices; i++) {
            CHECK_CUDA_ERRORS(hipSetDevice(i));
            CHECK_CUDA_ERRORS(hipStreamCreate(&stream[i]));

            // Allocate device memory for rays and copy from host to device
            CHECK_CUDA_ERRORS(hipMalloc(&d_rays[i], sizeof(GPURay) * raysPerDevice));
            CHECK_CUDA_ERRORS(hipMemcpyAsync(d_rays[i], rayons.data() + i * raysPerDevice, sizeof(GPURay) * raysPerDevice, hipMemcpyHostToDevice, stream[i]));

            CHECK_CUDA_ERRORS(hipMalloc(&d_results[i], sizeof(int) * raysPerDevice));

            d_tree[i] = M_root_gpu_tree->DeepCopy(M_root_gpu_tree);

            // Launch the kernel with one block per ray
            GPU_traverse_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_tree[i], d_rays[i], d_results[i], raysPerDevice);

            // Copy back results, the first fustrum wil be place from results[0] to results[raysPerDevice - 1] and so on
            CHECK_CUDA_ERRORS(hipMemcpyAsync(results.data() + i * raysPerDevice, d_results[i], sizeof(int) * raysPerDevice, hipMemcpyDeviceToHost, stream[i]));
        }

        for (int i = 0; i < numDevices; i++) 
        {
            CHECK_CUDA_ERRORS(hipSetDevice(i));
            CHECK_CUDA_ERRORS(hipStreamSynchronize(stream[i]));
            CHECK_CUDA_ERRORS(hipStreamDestroy(stream[i]));
            CHECK_CUDA_ERRORS(hipFree(d_tree[i]));
            CHECK_CUDA_ERRORS(hipFree(d_rays[i]));
            CHECK_CUDA_ERRORS(hipFree(d_results[i]));
        }

        return results;
    }
};

// Wrapper to be able to call the GPU function from the CPU
std::vector<int> GPUraySearchWrapper(std::vector<Feel::BVHRay> const& rays, const Feel::BVHTree<3> * tree)
{
    return GPUBVH::GPUraySearch(rays, tree);
}
